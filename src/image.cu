#include "hip/hip_runtime.h"
#include "image.h"
#include <iostream>

Image::Image(){
}

Image::Image(size_t m, size_t n){
  allocate_pixel_memory(m, n, false);
}

Image::Image(size_t m, size_t n, bool pinned){
  allocate_pixel_memory(m, n, pinned);
}

Image::~Image(){
  for(size_t i = 0; i < m+2*BORDER_SIZE; i++){
    if(is_pinned)
      hipFree(pixels[i]);
    else
      free(pixels[i]);
  }

  if(is_pinned)
    hipFree(pixels);
  else
    free(pixels);
}

void Image::allocate_pixel_memory(size_t m, size_t n, bool pinned){
  this->m = m;
  this->n = n;

  is_pinned = pinned;

  if(pinned)
    hipHostMalloc(&pixels, sizeof(float*)*(m+BORDER_SIZE*2));
  else
    pixels = (float**)aligned_alloc(ALIGNMENT, sizeof(float*)*(m+BORDER_SIZE*2));

  for(size_t i = 0; i < m+2*BORDER_SIZE; i++){
    if(pinned)
      hipHostMalloc(&pixels[i], sizeof(float*)*(n+BORDER_SIZE*2));
    else
      pixels[i] = (float*)aligned_alloc(ALIGNMENT, sizeof(float)*(n+BORDER_SIZE*2));

    if (i < BORDER_SIZE || i >= m){
      for(size_t j = 0; j < n+2*BORDER_SIZE; j++){
        pixels[i][j] = BORDER_VALUE;
      }
    }
    else {
      for(size_t j = 0; j < BORDER_SIZE; j++){
        pixels[i][j] = BORDER_VALUE;
      }
      for(size_t j = n; j < n+BORDER_SIZE; j++){
        pixels[i][j] = BORDER_VALUE;
      }
    }
  }
}

size_t Image::get_m(){ return m; }
size_t Image::get_height(){ return m; }
size_t Image::get_n(){ return n; }
size_t Image::get_width(){ return n; }

bool Image::operator==(Image& other){
  if(this->get_m() != other.get_m() || this->get_n() != other.get_n()){
    return false;
  }

  for(size_t i = 0; i < m; i++){
    for(size_t j = 0; j < n; j++){
      if (*this->at(i, j) != *other.at(i, j)) {
        return false;
      }
    }
  }
  
  return true;
}

bool Image::operator!=(Image& other){ return !(*this == other); }

float* Image::at(size_t i, size_t j){
  return &pixels[i+BORDER_SIZE][j+BORDER_SIZE];
}

Image Image::load_image(string filename){
  ifstream input;
  input.open(filename);
  string line;
  std::size_t current, previous;
  string delim = " ";

  // skip first line, we just support P2
  getline(input, line);

  // second line has width and height
  getline(input, line);
  auto i = line.find(delim);
  size_t width = stoi(line.substr(0, i));
  auto j = line.find(delim);
  size_t height = stoi(line.substr(i+1, j));

  // skip third line, we just support 255 as max value
  getline(input, line);

  string image_string;
  while(getline(input, line)){
    image_string += line;
  }

  Image image(height, width);
  current = 0;
  previous = 0;

  for (size_t i = 0; i < height; i++){
    for (size_t j = 0; j < width; j++){
      current = image_string.find(delim, previous);
      (*image.at(i,j)) = stoi(image_string.substr(previous, current-previous));
      previous = current + 1;
    }
  }

  input.close();

  return image;
}

void Image::save_image(Image &image, string filename){
  ofstream output;
  output.open(filename);
  string row_output;

  // outputing header
  output << "P2" << endl;
  // n is width, m is height
  output << to_string(image.get_n()) << " "
         << to_string(image.get_m()) << endl;
  output << "255" << endl;

  for(size_t i = 0; i < image.get_m(); i++){
    row_output = "";
    for(size_t j = 0; j < image.get_n(); j++){
      row_output += to_string(uint8_t(*image.at(i,j))) + " ";
    }
    output << row_output << endl;
  }

  output.close();
}

// m is height, n is length
void Image::randomize(){
  for(size_t i = 0; i < m; i++){
    for(size_t j = 0; j < n; j++){
      *at(i, j) = rand() % 256;
    }
  }
}

void Image::copy_to_gpu(){
  is_on_gpu = true;

  vector<hipError_t> errors;

  gpu_pixels = (float**)aligned_alloc(ALIGNMENT, sizeof(float*)*m);

  for (size_t i = 0; i < m; i++){
    errors.push_back(hipMalloc(&gpu_pixels[i], 
      sizeof(float)*(n+2*BORDER_SIZE)));
    errors.push_back(hipMemcpy(gpu_pixels[i], pixels[i],
      sizeof(float)*(n+2*BORDER_SIZE), hipMemcpyDefault));
  }

  errors.push_back(hipMalloc(&d_gpu_pixels, sizeof(float*)*m));
  errors.push_back(hipMemcpy(d_gpu_pixels, gpu_pixels, sizeof(float*)*m,
                              hipMemcpyDefault));

  for (size_t i = 0; i < errors.size(); i++){
    if(errors[i] != 0){
      cerr << "error number " << i << ": " 
           << hipGetErrorName(errors[i]) << endl
           << hipGetErrorString(errors[i]) << endl;
    }
  }
}

void Image::copy_from_gpu(){
  vector<hipError_t> errors;

  for (size_t i = 0; i < m; i++){
    errors.push_back(hipMemcpy(pixels[i], gpu_pixels[i],
      sizeof(float)*(n+2*BORDER_SIZE), hipMemcpyDefault));
  }

  for (size_t i = 0; i < errors.size(); i++){
    if(errors[i] != 0){
      cerr << "error number " << i << ": " 
           << hipGetErrorName(errors[i]) << endl
           << hipGetErrorString(errors[i]) << endl;
    }
  }
}

