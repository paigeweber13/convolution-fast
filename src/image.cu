#include "hip/hip_runtime.h"
#include "image.h"
#include <iostream>

Image::Image(){
}

Image::Image(size_t m, size_t n){
  allocate_pixel_memory(m, n, false);
}

Image::Image(size_t m, size_t n, bool pinned){
  allocate_pixel_memory(m, n, pinned);
}

Image::~Image(){
  for(size_t i = 0; i < m+2*BORDER_SIZE; i++){
    free(pixels[i]);
  }

  free(pixels);
}

void Image::allocate_pixel_memory(size_t m, size_t n, bool pinned){
  this->m = m;
  this->n = n;

  pixels = (float**)aligned_alloc(ALIGNMENT, sizeof(float*)*(m+BORDER_SIZE*2));

  for(size_t i = 0; i < m+2*BORDER_SIZE; i++){
    pixels[i] = (float*)aligned_alloc(ALIGNMENT, sizeof(float)*(n+BORDER_SIZE*2));

    if (i < BORDER_SIZE || i >= m){
      for(size_t j = 0; j < n+2*BORDER_SIZE; j++){
        pixels[i][j] = BORDER_VALUE;
      }
    }
    else {
      for(size_t j = 0; j < BORDER_SIZE; j++){
        pixels[i][j] = BORDER_VALUE;
      }
      for(size_t j = n; j < n+BORDER_SIZE; j++){
        pixels[i][j] = BORDER_VALUE;
      }
    }
  }
}

size_t Image::get_m(){ return m; }
size_t Image::get_height(){ return m; }
size_t Image::get_n(){ return n; }
size_t Image::get_width(){ return n; }

bool Image::operator==(Image& other){
  if(this->get_m() != other.get_m() || this->get_n() != other.get_n()){
    return false;
  }

  for(size_t i = 0; i < m; i++){
    for(size_t j = 0; j < n; j++){
      if (*this->at(i, j) != *other.at(i, j)) {
        return false;
      }
    }
  }
  
  return true;
}

bool Image::operator!=(Image& other){ return !(*this == other); }

float* Image::at(size_t i, size_t j){
  return &pixels[i+BORDER_SIZE][j+BORDER_SIZE];
}

Image Image::load_image(string filename){
  ifstream input;
  input.open(filename);
  string line;
  std::size_t current, previous;
  string delim = " ";

  // skip first line, we just support P2
  getline(input, line);

  // second line has width and height
  getline(input, line);
  auto i = line.find(delim);
  size_t width = stoi(line.substr(0, i));
  auto j = line.find(delim);
  size_t height = stoi(line.substr(i+1, j));

  // skip third line, we just support 255 as max value
  getline(input, line);

  string image_string;
  while(getline(input, line)){
    image_string += line;
  }

  Image image(height, width);
  current = 0;
  previous = 0;

  for (size_t i = 0; i < height; i++){
    for (size_t j = 0; j < width; j++){
      current = image_string.find(delim, previous);
      (*image.at(i,j)) = stoi(image_string.substr(previous, current-previous));
      previous = current + 1;
    }
  }

  input.close();

  return image;
}

void Image::save_image(Image &image, string filename){
  ofstream output;
  output.open(filename);
  string row_output;

  // outputing header
  output << "P2" << endl;
  // n is width, m is height
  output << to_string(image.get_n()) << " "
         << to_string(image.get_m()) << endl;
  output << "255" << endl;

  for(size_t i = 0; i < image.get_m(); i++){
    row_output = "";
    for(size_t j = 0; j < image.get_n(); j++){
      row_output += to_string(uint8_t(*image.at(i,j))) + " ";
    }
    output << row_output << endl;
  }

  output.close();
}

// m is height, n is length
void Image::randomize(){
  for(size_t i = 0; i < m; i++){
    for(size_t j = 0; j < n; j++){
      *at(i, j) = rand() % 256;
    }
  }
}

