#include "hip/hip_runtime.h"
#include "convolution_gpu.h"

void convolve_gpu(Image &input, Image &output, Kernel &kernel){
  convolve_gpu(input, output, kernel, input.get_m(), input.get_n());
}

void convolve_gpu(Image &input, Image &output, Kernel &kernel,
    size_t num_blocks, size_t num_threads){
  printf("copying input...\n");
  input.copy_to_gpu();
  printf("copying output...\n");
  output.copy_to_gpu();
  printf("convolving...\n");
  d_convolve_gpu<<<num_blocks, num_threads>>>(input.d_gpu_pixels, 
      output.d_gpu_pixels, kernel.d_gpu_values, input.get_m(), input.get_n(),
      kernel.get_k());
  printf("copying back result...\n");
  output.copy_from_gpu();
}

void __global__ d_convolve_gpu(float** d_input, float** d_output, 
    float** d_kernel, size_t m, size_t n, size_t k){
  size_t y = blockIdx.x + BORDER_SIZE;
  size_t x = threadIdx.x + BORDER_SIZE;
  
  float sum = 0;
  for(size_t o = 0; o < k; o++){
    for(size_t p = 0; p < k; p++){
      sum += d_input[y-m+o][x-m+p] * d_kernel[o][p];
    }
  }
  d_output[y][x] = uint8_t(sum);
}

// copy async data in
// run kernel in stream
// copy async data out

