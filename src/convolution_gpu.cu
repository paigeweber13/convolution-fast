#include "hip/hip_runtime.h"
#include "convolution_gpu.h"

void convolve_gpu(Image input, Image output, Kernel kernel){
  convolve_gpu(input, output, kernel, input.get_m(), input.get_n());
}

void convolve_gpu(Image input, Image output, Kernel kernel, size_t num_blocks,
    size_t num_threads){
  input.copy_to_gpu();
  output.copy_to_gpu();
  d_convolve_gpu<<<num_blocks, num_threads>>>(input.d_gpu_pixels, 
      output.d_gpu_pixels, kernel.d_gpu_values, input.get_m(), input.get_n(),
      kernel.get_k());
  output.copy_from_gpu();
}

void __global__ d_convolve_gpu(float** d_input, float** d_output, 
    float** d_kernel, size_t m, size_t n, size_t k){
  size_t y = blockIdx.x;
  size_t x = threadIdx.x;
  
  float sum = 0;
  for(size_t o = 0; o < k; o++){
    for(size_t p = 0; p < k; p++){
      sum += d_input[y-m+o][x-m+p] * d_kernel[o][p];
    }
  }
  d_output[y][x] = uint8_t(sum);
}

// copy async data in
// run kernel in stream
// copy async data out

