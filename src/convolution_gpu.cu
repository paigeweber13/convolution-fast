#include "hip/hip_runtime.h"
#include "convolution_gpu.h"

void convolve_gpu(Image &input, Image &output, Kernel &kernel){
  convolve_gpu(input, output, kernel, input.get_m(), 32);
}

void convolve_gpu(Image &input, Image &output, Kernel &kernel,
    size_t num_blocks, size_t num_threads){
  input.copy_to_gpu();
  output.copy_to_gpu();
  for(size_t i = BORDER_SIZE; i < input.get_n() + BORDER_SIZE; i += num_threads){
    // printf("i: %lu\n", i);
    d_convolve_gpu<<<num_blocks, num_threads>>>(input.d_gpu_pixels, 
        output.d_gpu_pixels, kernel.d_gpu_values, input.get_m(), input.get_n(),
        kernel.get_k(), kernel.get_midpoint(), i);
  }
  output.copy_from_gpu();
}

void __global__ d_convolve_gpu(float** d_input, float** d_output, 
    float** d_kernel, size_t m, size_t n, size_t k, int mid, size_t start_i){
  size_t x = start_i + threadIdx.x + BORDER_SIZE;
  size_t y = blockIdx.x + BORDER_SIZE;
  if(start_i > 7){
    printf("start_i: %lu\n", start_i);
  }
  // printf("x, y: %lu, %lu\n", x, y);
  if(y == 500 && x == 500){
    printf("inside device code\n");
    printf("input 500, 500: %f\n", d_input[500][500]);
  }
  
  if(x < n && y < m){
    float sum = 0;
    for(size_t o = 0; o < k; o++){
      for(size_t p = 0; p < k; p++){
        sum += d_input[y-mid+o][x-mid+p] * d_kernel[o][p];
      }
    }
    d_output[y][x] = uint8_t(sum);
  }

  if(y == 500 && x == 500){
    printf("output 500, 500: %f\n", d_output[500][500]);
  }
}

// copy async data in
// run kernel in stream
// copy async data out

