#include "hip/hip_runtime.h"
#include "kernel.h"

#include <iostream>

Kernel::Kernel(){
  Kernel(3);
}

Kernel::Kernel(size_t k){
  if (k < 3){
    throw "Unusable k. K must be at least 3.";
  } else if (k % 2 == 0){
    throw "Unusable k. K must be odd.";
  } 

  // cout << "allocating kernel..." << endl;
  values = (float **)aligned_alloc(ALIGNMENT, k*sizeof(float *));
  is_on_gpu = false;

  for (size_t i = 0; i < k; i++){
    // cout << "allocating kernel row " << i << endl;
    values[i] = (float *)aligned_alloc(ALIGNMENT, k*sizeof(float));
  }

  this->k = k;
  midpoint = int(ceil(k/2));
  // cout << "k: " << this->k << endl;
  // cout << "midpoint: " << this->midpoint << endl;
}

Kernel::~Kernel(){
  // cout << "freeing kernel..." << endl;
  for (size_t i = 0; i < k; i++){
    // printf("freeing kernel row %lu\n", i);
    free(values[i]);
  }

  // printf("freeing rest of kernel \n");
  free(values);

  if (is_on_gpu) {
    for (size_t i = 0; i < k; i++){
      hipFree(gpu_values[i]);
    }

    hipFree(d_gpu_values);
    free(gpu_values);
  }
}

int Kernel::get_midpoint(){ return midpoint; }
size_t Kernel::get_k(){ return k; }

bool Kernel::operator==(Kernel& other){
  if(this->get_k() != other.get_k()){
    return false;
  }

  for(size_t i = 0; i < k; i++){
    for(size_t j = 0; j < k; j++){
      if (this->values[i][j] != other.values[i][j]) {
        return false;
      }
    }
  }
  
  return true;
}

bool Kernel::operator!=(Kernel& other){ return !(*this == other); }

string Kernel::to_string(){
  stringstream ss;
  ss << "k: " << k << endl;
  ss << "Midpoint: " << midpoint << endl;
  ss << "Kernel:" << endl;
  for(size_t i = 0; i < k; i++){
    for(size_t j = 0; j < k; j++){
      ss << fixed << setprecision(4) << values[i][j] << "  ";
    }
    ss << endl;
  }
  ss << endl;
  return ss.str();
}

float Kernel::bivariate_gaussian(float std_dev, float x, float y){
  return 1/(2*M_PI*pow(std_dev, 2)) * 
    exp(-(pow(x, 2) + pow(y, 2))/(2*pow(std_dev,2)));
}

void Kernel::make_blur_kernel(){
  // will need tuning
  // float std_dev = float(k) * 2/3;
  float std_dev = 2.0;

  float sum = 0.0;
  for(int x = -midpoint; x <= midpoint; x++){
    for(int y = -midpoint; y <= midpoint; y++){
      auto current_value = bivariate_gaussian(std_dev, x, y);
      values[x+midpoint][y+midpoint] = current_value;
      sum += current_value;
    }
  }

  // normalize!
  for(size_t i = 0; i < get_k(); i++){
    for(size_t j = 0; j < get_k(); j++){
      values[i][j] /= sum;
    }
  }
}

void Kernel::copy_to_gpu(){
  is_on_gpu = true;

  vector<hipError_t> errors;

  gpu_values = (float**)aligned_alloc(ALIGNMENT, sizeof(float*)*k);

  for (size_t i = 0; i < k; i++){
    errors.push_back(hipMalloc(&gpu_values[i], sizeof(float)*k));
    errors.push_back(hipMemcpy(gpu_values[i], values[i], sizeof(float)*k,
                                hipMemcpyDefault));
  }

  errors.push_back(hipMalloc(&d_gpu_values, sizeof(float*)*k));
  errors.push_back(hipMemcpy(d_gpu_values, gpu_values, sizeof(float*)*k,
                              hipMemcpyDefault));

  for (size_t i = 0; i < errors.size(); i++){
    if(errors[i] != 0){
      cerr << "error number " << i << ": " 
           << hipGetErrorName(errors[i]) << endl
           << hipGetErrorString(errors[i]) << endl;
    }
  }
}

Kernel Kernel::generate_blur_kernel(size_t k){
  Kernel result(k);
  // will need tuning
  // float std_dev = float(k) * 2/3;
  float std_dev = 2.0;

  float sum = 0.0;
  for(int x = -result.get_midpoint(); x <= result.get_midpoint(); x++){
    for(int y = -result.get_midpoint(); y <= result.get_midpoint(); y++){
      auto current_value = bivariate_gaussian(std_dev, x, y);
      result.values[x+result.get_midpoint()][y+result.get_midpoint()]
        = current_value;
      sum += current_value;
    }
  }

  // normalize!
  for(size_t i = 0; i < result.get_k(); i++){
    for(size_t j = 0; j < result.get_k(); j++){
      result.values[i][j] /= sum;
    }
  }

  return result;
}

vector<Kernel> Kernel::generate_blur_kernels(size_t max_k){
  vector<Kernel> result(max_k+1);
  for(size_t i = 3; i <= max_k; i += 2){
    result[i] = generate_blur_kernel(i);
  }
  return result;
}

